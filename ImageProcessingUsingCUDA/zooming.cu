#include "hip/hip_runtime.h"
//This program made by Apoorva Chauhan

//This program is working for PPM (P3) format images
//PPM has RGB values as input data

/* The demo file is as follows..
P3
# The P3 means colors are in ASCII, then 3 columns and 2 rows, then 255 for max color, then RGB triplets-----  THIS COMMENT IS OPTIONAL
3 2
255
255   0   0     0 255   0     0   0 255
255 255   0   255 255 255     0   0   0
*/

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>

	
__global__ void copy( int *a, int *b, int col1, int row1, int col, int row, int factor )
{
        int tx = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
        int ty = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	if(tx<row && ty<col1)
	{
		i1=ty*factor;
		j1=tx*factor;
		b[(i1*col1*3) + (j1*3)] = a[(ty*col*3) + (tx*3)];
		b[(i1*col1*3) + (j1*3 + 1)] = a[(ty*col*3) + (tx*3 + 1)];
		b[(i1*col1*3) + (j1*3 + 2)] = a[(ty*col*3) + (tx*3 + 2)];
	}
}

__global__ void hori_inter( int *a, int *b, float si, float co, int col, int row )
{
	int idx = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
        int idy = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	i1=0;
	while(i1<row1)
	{
		j1=0;
		while(j1<(col1-factor))
		{
			x1 = j1;
			x2 = j1+factor;
			for(j=1;j<factor;j++)
			{
				x = j1+j;
				b[(i1*col1*3) + (x*3)] = ((x2-x)*b[(i1*col1*3) + (x1*3)]/(x2-x1)) + ((x-x1)*b[(i1*col1*3) + (x2*3)]/(x2-x1));
                		b[(i1*col1*3) + (x*3 + 1)] = ((x2-x)*b[(i1*col1*3) + ((x1*3) + 1)]/(x2-x1)) + ((x-x1)*b[(i1*col1*3) + ((x2*3) + 1)]/(x2-x1));
                		b[(i1*col1*3) + (x*3 + 2)] = ((x2-x)*b[(i1*col1*3) + ((x1*3) + 2)]/(x2-x1)) + ((x-x1)*b[(i1*col1*3) + ((x2*3) + 2)]/(x2-x1));
			}
			j1+=factor;
		}
		i1+=factor;
	}
}

__global__ void ver_inter( int *a, int *b, float si, float co, int col, int row )
{
        int idx = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
        int idy = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	i1=0;
        while(i1<(row1-factor))
        {
                x1=i1;
		x2=i1 + factor;       
                for(j=1;j<factor;j++)
                {
                	x = i1+j;
			j1=0;
			while(j1<col1)
			{
                        b[(x*col1*3) + (j1*3)] = ((x1-x)*b[(x2*col1*3) + (j1*3)]/(x1-x2)) + ((x-x2)*b[(x1*col1*3) + (j1*3)]/(x1-x2));
                        b[(x*col1*3) + (j1*3 + 1)] = ((x1-x)*b[(x2*col1*3) + ((j1*3) + 1)]/(x1-x2)) + ((x-x2)*b[(x1*col1*3) + ((j1*3) + 1)]/(x1-x2));
                        b[(x*col1*3) + (j1*3 + 2)] = ((x1-x)*b[(x2*col1*3) + ((j1*3) + 2)]/(x1-x2)) + ((x-x2)*b[(x1*col1*3) + ((j1*3) + 2)]/(x1-x2));
                       	j1++;
			}
		}
                i1+=factor;
        }
}

int main()
{
    FILE *fp,*fp1;
    fp = fopen("blackbuck.ppm","r");
    fp1 = fopen("zoomed_image.ppm","w");
    char c;
    int col=0,row=0,max=0,i=0,j=0;
    float degree=0.0f,co=0.0f,si=0.0f;
    int *a,*b,*a_d,*b_d;
	hipEvent_t start,stop;
        float time;
    
    //The degree of rotation is entered in radians
	printf("\n\nEnter the degree of rotation\t");
	scanf("%f",&degree);
	    
    if(fp == NULL)
    {
          printf("\nSource File does not exist...");
          exit(1);
    }
    else
    {
        //This step is to skip the image type specified, like P3
        
        c=fgetc(fp);
        fputc(c,fp1);
        c=fgetc(fp);
        fputc(c,fp1);
        c=fgetc(fp);
        fputc(c,fp1);
        
        fscanf(fp,"%d",&col);
        c = fgetc(fp);
        
        //This step is done to skip the statements, if present in comment '#'
        
        while( c == '#')
        {
                fputc(c,fp1);
                c=fgetc(fp);
                fputc(c,fp1);
                while(c != '\n')
                {
                        c=fgetc(fp);
                        fputc(c,fp1);
                }
                fscanf(fp,"%d",&col);
                c=fgetc(fp);
        }
        
        fscanf(fp,"%d",&row);
        fscanf(fp,"%d",&max);        
        fprintf(fp1,"%d",col);
        fprintf(fp1,"%c",' ');
	
	printf("\nEnter the scaling factor :\t");
        scanf("%d",&factor);
        row1 = (factor*row) - (factor-1);
        col1 = (factor*col) - (factor-1);

        fprintf(fp1,"%d",col1);
        fprintf(fp1,"%c",' ');
	fprintf(fp1,"%d",row1);
        fprintf(fp1,"%c",'\n');
        fprintf(fp1,"%d",max);
        fprintf(fp1,"%c",'\n');

	size_t size_a = sizeof(int)*row*col*3;
	size_t size_b = sizeof(int)*row1*col1*3;
	a = (int*)malloc(size_a);
        hipMalloc( (void**) &a_d, size_a);
        b = (int*)malloc(size_ab);
        hipMalloc( (void**) &b_d, size_b);
	hipMemset(b_d,0,size_b);

	printf("\nDegree %f",degree);
	co=cos(degree);
	si=sin(degree);
	printf("\n\n%f  %f\n\n",si,co);
	        
        for(i=0;i<row;i++)
        {
            for(j=0;j< col*3;j++)
            {
                fscanf(fp,"%d",&a[(i*col*3)+j]);
	    }
        }
                
	hipMemcpy(a_d,a,size_ab,hipMemcpyHostToDevice);
	
	hipEventCreate(&start);
 	hipEventCreate(&stop);
	hipEventRecord(start,0);
		
	dim3 dimBlock(16,16);
	dim3 dimGrid( ((col-1)/dimBlock.x)+1,((row-1)/dimBlock.y)+1  );
	
	for(i=0;i<10;i++)
	{
	copy<<<dimGrid,dimBlock>>>(a_d,b_d,col1,row1,col,row,factor);
	hipDeviceSynchronize();
	hori_inter<<<dimGrid,dimBlock>>>(a_d,b_d,col1,row1,col,row);
	hipDeviceSynchronize();
	ver_inter<<<dimGrid,dimBlock>>>(a_d,b_d,col1,row1,col,row);
	hipDeviceSynchronize();
	}
	
        printf("\nKernel error : %s", hipGetErrorString(hipGetLastError())); 
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time,start,stop);
	
	hipMemcpy(b,b_d,size_ab,hipMemcpyDeviceToHost);
	printf("\nMemcpy error : %s", hipGetErrorString(hipGetLastError()));	
	for(i=0;i<row;i++)
        {
            for(j=0;j< col*3;j++)
            {         
                fprintf(fp1,"%d",b[(i*col*3)+j]);
                fprintf(fp1,"%c",' ');
            }
            fprintf(fp1,"%c",'\n');
        }
    }

    	printf("\n\nProcessing time is:\t%f (ms)\n\n",time/10);
        hipEventDestroy(start);
        hipEventDestroy(stop);
	
	fclose(fp);
    	fclose(fp1);
    	free(a);
    	free(b);
    	hipFree(a_d);
	hipFree(b_d);
	return 0;
}		
