#include "hip/hip_runtime.h"
//This program made by Apoorva Chauhan

//This program is working for PPM (P3) format images
//PPM has RGB values as input data

/* The demo file is as follows..
P3
# The P3 means colors are in ASCII, then 3 columns and 2 rows, then 255 for max color, then RGB triplets-----  THIS COMMENT IS OPTIONAL
3 2
255
255   0   0     0 255   0     0   0 255
255 255   0   255 255 255     0   0   0
*/

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>
#include<stdlib.h>

__global__ void rotation( int *a, int *b, float si, float co, int col, int row )
{
	int idx = __umul24(blockIdx.x,blockDim.x) + threadIdx.x;
	int idy = __umul24(blockIdx.y,blockDim.y) + threadIdx.y;
	float i_r=0.0f,j_r=0.0f;
	int i_temp=0,j_temp=0;

	if(idx<col && idy<row)
	{
		i_r = co*(idy-((row-1)/2))-si*(idx-((col-1)/2))+((row-1)/2);
                j_r = si*(idy-((row-1)/2))+co*(idx-((col-1)/2))+((col-1)/2);
                i_temp = i_r;
                j_temp = j_r;
                if( (i_r-i_temp) > 0.5)
        	        i_temp++;
                if( (j_r-j_temp) > 0.5)
                        j_temp++;
				
		if(i_temp < row && i_temp >= 0 && j_temp < col && j_temp >=0)
		{
                        
 b[__umul24(i_temp,__umul24(col,3)) + __umul24(j_temp,3)]       = a[(__umul24(idy,__umul24(col,3))) + __umul24(idx,3)];
 b[__umul24(i_temp,__umul24(col,3)) + (__umul24(j_temp,3) + 1)] = a[__umul24(idy,__umul24(col,3)) + (__umul24(idx,3) + 1)];
 b[__umul24(i_temp,__umul24(col,3)) + (__umul24(j_temp,3) + 2)] = a[__umul24(idy,__umul24(col,3)) + (__umul24(idx,3) + 2)];
		}
	}
}

int main()
{
    FILE *fp,*fp1;
    fp = fopen("blackbuck.ppm","r");
    fp1 = fopen("rotated_image.ppm","w");
    char c;
    int col=0,row=0,max=0,i=0,j=0;
    float degree=0.0f,co=0.0f,si=0.0f;
    int *a,*b,*a_d,*b_d;
	hipEvent_t start,stop;
        float time;
    
    //The degree of rotation is entered in radians
	printf("\n\nEnter the degree of rotation\t");
	scanf("%f",&degree);
	    
    if(fp == NULL)
    {
          printf("\nSource File does not exist...");
          exit(1);
    }
    else
    {
        //This step is to skip the image type specified, like P3
        
        c=fgetc(fp);
        fputc(c,fp1);
        c=fgetc(fp);
        fputc(c,fp1);
        c=fgetc(fp);
        fputc(c,fp1);
        
        fscanf(fp,"%d",&col);
        c = fgetc(fp);
        
        //This step is done to skip the statements, if present in comment '#'
        
        while( c == '#')
        {
                fputc(c,fp1);
                c=fgetc(fp);
                fputc(c,fp1);
                while(c != '\n')
                {
                        c=fgetc(fp);
                        fputc(c,fp1);
                }
                fscanf(fp,"%d",&col);
                c=fgetc(fp);
        }
        
        fscanf(fp,"%d",&row);
        fscanf(fp,"%d",&max);        
        fprintf(fp1,"%d",col);
        fprintf(fp1,"%c",' ');
        fprintf(fp1,"%d",row);
        fprintf(fp1,"%c",'\n');
        fprintf(fp1,"%d",max);
        fprintf(fp1,"%c",'\n');

	size_t size_ab = sizeof(int)*row*col*3;
	a = (int*)malloc(size_ab);
        hipMalloc( (void**) &a_d, size_ab);
        b = (int*)malloc(size_ab);
        hipMalloc( (void**) &b_d, size_ab);
	hipMemset(b_d,0,size_ab);

	printf("\nDegree %f",degree);
	co=cos(degree);
	si=sin(degree);
	printf("\n\n%f  %f\n\n",si,co);
	        
        for(i=0;i<row;i++)
        {
            for(j=0;j< col*3;j++)
            {
                fscanf(fp,"%d",&a[(i*col*3)+j]);
		}
        }
                
	hipMemcpy(a_d,a,size_ab,hipMemcpyHostToDevice);
	
	hipEventCreate(&start);
 	hipEventCreate(&stop);
	hipEventRecord(start,0);
		
	dim3 dimBlock(16,16);
	dim3 dimGrid( ((col-1)/dimBlock.x)+1,((row-1)/dimBlock.y)+1  );
	
	for(i=0;i<10;i++)
	{
	rotation<<<dimGrid,dimBlock>>>(a_d,b_d,si,co,col,row);
	hipDeviceSynchronize();
	}
        printf("\nKernel error : %s", hipGetErrorString(hipGetLastError())); 
        hipEventRecord(stop,0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time,start,stop);
	
	hipMemcpy(b,b_d,size_ab,hipMemcpyDeviceToHost);
	printf("\nMemcpy error : %s", hipGetErrorString(hipGetLastError()));	

    //averaging filter
    
		for(i=0;i<row;i++)
	{
		for(j=0;j<col;j++)
		{
			x1=i-1;
			y1=j-1;
			x2=i+1;
			y2=j+1;
						
			if(i>0 &&i<row-1 && j>0 && j<col-1)
			{
				b[(i*col*3) + (j*3)] = (b[(i*col*3) + (y1*3)]+b[(x2*col*3) + (j*3)]+b[(i*col*3) + (y2*3)]+b[(x1*col*3) + (j*3)])/4;
                b[(i*col*3) + (j*3 + 1)] = (b[(i*col*3) + (y1*3+1)]+b[(x2*col*3) + (j*3+1)]+b[(i*col*3) + (y2*3+1)]+b[(x1*col*3) + (j*3+1)])/4;
                b[(i*col*3) + (j*3 + 2)] =  (b[(i*col*3) + (y1*3+2)]+b[(x2*col*3) + (j*3+2)]+b[(i*col*3) + (y2*3+2)]+b[(x1*col*3) + (j*3+2)])/4;

			}

		}
	}


	for(i=0;i<row;i++)
        {
            for(j=0;j< col*3;j++)
            {         
                fprintf(fp1,"%d",b[(i*col*3)+j]);
                fprintf(fp1,"%c",' ');
            }
            fprintf(fp1,"%c",'\n');
        }
    }

    	printf("\n\nProcessing time is:\t%f (ms)\n\n",time/10);
        hipEventDestroy(start);
        hipEventDestroy(stop);
	
	fclose(fp);
    	fclose(fp1);
    	free(a);
    	free(b);
    	hipFree(a_d);
	hipFree(b_d);
	return 0;
}		
